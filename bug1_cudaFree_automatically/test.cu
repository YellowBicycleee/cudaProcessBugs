
#include <hip/hip_runtime.h>
#include <iostream>
#define CHECK_CUDA(cmd)                                                                                   \
  do {                                                                                                    \
    hipError_t err = cmd;                                                                                \
    if (err != hipSuccess) {                                                                             \
      fprintf(stderr, "CUDA error: %s, file %s, line %d\n", hipGetErrorString(err), __FILE__, __LINE__); \
      exit(1);                                                                                            \
    }                                                                                                     \
  } while (0)

struct Pointer {
	void* ptr;
	Pointer () : ptr (nullptr) {}
	void alloc(int size) {
		if (ptr) hipFree(ptr);
		CHECK_CUDA(hipMalloc(&ptr, size));
	}
	virtual ~Pointer() {
		if (ptr != nullptr) {
			CHECK_CUDA(hipFree(ptr));
		}
		printf("destructor\n");
	}
};

//Pointer *a;
Pointer a;  // when process end, destructor will report error CUDA error: driver shutting down
int main () {
	// Pointer p;
	// p.alloc(10000);
	a.alloc(10000);
	return 0;
}



